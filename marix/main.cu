#include <stdio.h>
#include <stdlib.h>
#include "matrix.cuh"

// Define matrix dimensions
#define MATRIX_SIZE_ROW 32
#define MATRIX_SIZE_COL 32


int main()
{
    float *a, *b, *c;             // Host matrices
    float *dev_a, *dev_b, *dev_c; // Device matrices

    a = (float *)malloc(sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);
    b = (float *)malloc(sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);
    c = (float *)malloc(sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);

    // Allocate memory for host matrices
    FILE *filea = fopen("matrix_a.txt", "w");
    FILE *fileb = fopen("matrix_b.txt", "w");
    // Initialize host matrices with random values
    for (int i = 0; i < MATRIX_SIZE_ROW * MATRIX_SIZE_COL; ++i)
    {

        a[i] = (float)(rand() % 10);
        b[i] = (float)(rand() % 10);
        fprintf(filea, "%lf ", a[i]);
        fprintf(fileb, "%lf ", b[i]);
       if (i % MATRIX_SIZE_COL == 0 && i!=0)        {
            fprintf(filea, "\n");
            fprintf(fileb, "\n");
        }
    }
    fclose(filea);
    fclose(fileb);

    // Allocate memory for device matrices
    hipMalloc((void **)&dev_a, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);
    hipMalloc((void **)&dev_b, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);
    hipMalloc((void **)&dev_c, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW);
    // Copy host matrices to device matrices
    hipMemcpy(dev_a, a, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW, hipMemcpyHostToDevice);
    ;
    // Define block and grid sizes
    dim3 blockDim(MATRIX_SIZE_COL, MATRIX_SIZE_ROW);
    dim3 gridDim((MATRIX_SIZE_COL + blockDim.x - 1) / blockDim.x, (MATRIX_SIZE_ROW + blockDim.y - 1) / blockDim.y);

    // Launch kernel to multiply matrices
    matrixMul<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c, MATRIX_SIZE_ROW, MATRIX_SIZE_COL);

    // Copy result matrix from device to host
    hipMemcpy(c, dev_c, sizeof(float) * MATRIX_SIZE_COL * MATRIX_SIZE_ROW, hipMemcpyDeviceToHost);

    FILE *filec = fopen("matrix_c.txt", "w");

    for (int i = 0; i < MATRIX_SIZE_ROW*MATRIX_SIZE_COL; i++)
    {
        fprintf(filec, "%lf ", c[i]);
        // printf("c[%d]=%f",i,c[i]);
        if (i % MATRIX_SIZE_COL == 0 && i!=0)
        {
            fprintf(filec, "\n");
        }
    }
    fclose(filec);

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}